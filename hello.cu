#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"

//Kernel
__global__
void hello_world(void){
	printf("hello\n");
}


int main(){
	printf("Chamando hello!\n");
	hello_world << <3, 10>> >();
	hipDeviceSynchronize();
	return 0;
}
